#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDABlas.h>

namespace at { namespace native {

Tensor &addmv_impl_cuda(Tensor& result, const Tensor &self, const Tensor &mat, const Tensor &vec, Scalar beta_, Scalar alpha_) {
  auto r_stride = result.stride(0);
  auto vec_size = vec.size(0);
  auto vec_stride = vec.stride(0);

  if (mat.scalar_type() == kHalf || mat.scalar_type() == kBFloat16) {
    // Currently no Hgemv/SgemvEx in Cublas
    Tensor vec_as_matrix = vec.reshape({vec_size, 1}).contiguous();
    Tensor self_as_matrix = self.reshape({mat.size(0), 1}).contiguous();
    at::addmm_out(result, self_as_matrix, mat, vec_as_matrix, beta_, alpha_);
    result.resize_({result.size(0)});
    return result;
  }

  AT_DISPATCH_FLOATING_TYPES(mat.scalar_type(), "addmv_impl_cuda", [&] {
    auto beta = beta_.to<scalar_t>();
    auto alpha = alpha_.to<scalar_t>();
    if (mat.stride(0) == 1) {
      at::cuda::blas::gemv<scalar_t>('n',
        mat.size(0), mat.size(1), alpha, mat.data_ptr<scalar_t>(), mat.stride(1), vec.data_ptr<scalar_t>(),
        vec_stride, beta, result.data_ptr<scalar_t>(), r_stride);
    }
    else if (mat.stride(1) == 1) {
      at::cuda::blas::gemv<scalar_t>('t',
        mat.size(1), mat.size(0), alpha, mat.data_ptr<scalar_t>(), mat.stride(0),
        vec.data_ptr<scalar_t>(), vec_stride, beta, result.data_ptr<scalar_t>(), r_stride);
    }
    else {
      Tensor cmat = mat.contiguous();
      at::cuda::blas::gemv<scalar_t>('t',
          mat.size(1), mat.size(0), alpha, cmat.data_ptr<scalar_t>(), cmat.stride(0),
          vec.data_ptr<scalar_t>(), vec.stride(0), beta, result.data_ptr<scalar_t>(), r_stride);
    }

    // In hipblasSgemv, hipblasDgemv (x,0).mv(0) does not
    // handle beta, whereas hipblasSgemm, hipblasDgemm do for case where (x,0).mm(0,y).
    if (vec.size(0) == 0 && mat.size(0) != 0) {
      if (beta == scalar_t(0)) {
        result.zero_();
      } else if (beta != scalar_t(1)) {
        result.mul_(beta);
      }
    }
  });
  return result;
}

}} // namespace at::native
