#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <THC/THCNumerics.cuh>
#include <ATen/cuda/HIPContext.h>

namespace at { namespace native {

template <typename integer>
constexpr inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template<typename scalar_t, typename idx_t, typename BinaryOperation>
__device__ void binary_op_update(const scalar_t lhs, scalar_t& rhs, const idx_t lhs_idx, idx_t& rhs_idx, BinaryOperation binary_op) {
  if(!THCNumerics<scalar_t>::isnan(rhs) && (THCNumerics<scalar_t>::isnan(lhs) || !binary_op(rhs, lhs))) {
    rhs = lhs;
    rhs_idx = lhs_idx;
  }
}
/* Perform an inclusive scan along the innermost dimension of a tensor.
 *
 * - num_rows is the size of the flattened outer dimensions;
 * - row_size is the size of the innermost dimension;
 *
 * The outer dimensions of the tensor are considered as a single dimension, i.e. the tensor is
 * considered as having 'num_rows' rows of size 'row_size'.
 * Each thread block processes one or more sets of contiguous rows (processing multiple rows
 * per thread block is quicker than processing a single row, especially for short rows).
 */
template<typename scalar_t, int num_threads_x, int num_threads_y, class BinaryFunction>
__global__ void tensor_kernel_scan_innermost_dim_with_indices(const scalar_t *self_, scalar_t *values_, int64_t *indices_,
                                                int num_rows, int row_size,
                                                scalar_t init, BinaryFunction binary_op) {
  __shared__ scalar_t vbuf[num_threads_y][2 * num_threads_x];
  __shared__ int64_t ibuf[num_threads_y][2 * num_threads_x];
  scalar_t* row_buf = vbuf[threadIdx.y];
  int64_t* row_idx_buf = ibuf[threadIdx.y];

  for (int block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    int row = block_row + threadIdx.y;
    const scalar_t *row_self = self_ + row * row_size;
    scalar_t *row_values = values_ + row * row_size;
    int64_t *row_indices = indices_ + row * row_size;
    scalar_t block_total = init;
    int64_t block_idx_final = 0;
    // Perform scan on one block at a time, keeping track of the total value of
    // all blocks processed so far.
    for (int block_col = 0; block_col < row_size; block_col += 2 * num_threads_x) {
      // Load data into shared memory (two values per thread).
      int col1 = block_col + threadIdx.x;
      int col2 = block_col + num_threads_x + threadIdx.x;
      if (row < num_rows) {
        if (col1 < row_size) {
          row_buf[threadIdx.x] = row_self[col1];
          row_idx_buf[threadIdx.x] = col1;
        } else {
          row_buf[threadIdx.x] = init;
          // No need to set the index here as the value in init will never be selected
        }

        if (col2 < row_size) {
          row_buf[num_threads_x + threadIdx.x] = row_self[col2];
          row_idx_buf[num_threads_x + threadIdx.x] = col2;
        } else {
          row_buf[num_threads_x + threadIdx.x] = init;
          // No need to set the index here as the value in init will never be selected
        }

        // Add the total value of all previous blocks to the first value of this block.
        if (threadIdx.x == 0) {
          binary_op_update(block_total, row_buf[0], block_idx_final, row_idx_buf[0], binary_op);
        }
      }
      __syncthreads();

      // Parallel reduction (up-sweep).
      for (int s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < num_rows && threadIdx.x < s) {
          int offset = (2 * threadIdx.x + 1) * d - 1;
          binary_op_update(row_buf[offset], row_buf[offset + d], row_idx_buf[offset], row_idx_buf[offset + d], binary_op);
        }
        __syncthreads();
      }

      // Down-sweep.
      for (int s = 2, d = num_threads_x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < num_rows && threadIdx.x < s - 1) {
          int offset = 2 * (threadIdx.x + 1) * d - 1;
          binary_op_update(row_buf[offset], row_buf[offset + d], row_idx_buf[offset], row_idx_buf[offset + d], binary_op);
        }
        __syncthreads();
      }

      // Write back to output.
      if (row < num_rows) {
        if (col1 < row_size){
          row_values[col1] = row_buf[threadIdx.x];
          row_indices[col1] = row_idx_buf[threadIdx.x];
        }
        if (col2 < row_size) {
          row_values[col2] = row_buf[num_threads_x + threadIdx.x];
          row_indices[col2] = row_idx_buf[num_threads_x + threadIdx.x];
        }
      }
      block_total = row_buf[2 * num_threads_x - 1];
      block_idx_final = row_idx_buf[2 * num_threads_x - 1];
      __syncthreads();
    }
  }
}

/* Perform an inclusive scan along an outer dimension of a tensor.
 *
 * - num_orows is the size of the flattened outer dimensions;
 * - num_irows is the size of the flattened inner dimensions;
 * - row_size is the size of the dimension along which to compute the variance;
 *
 * The dimensions to the outside and inside of the specified dimension are considered as flattened.
 * Thread blocks with the same blockIdx.y process an "outer row" (i.e. an element of the flattened
 * outer dimensions, which contains several "inner rows").
 * Each thread processes a single inner row at a time.
 */
template<typename scalar_t, class BinaryFunction>
__global__ void tensor_kernel_scan_outer_dim_with_indices(scalar_t *self_, scalar_t *values_, int64_t *indices_,
                  int num_orows, int num_irows, int row_size, scalar_t init, BinaryFunction binary_op) {
  for (int orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (int irow = blockIdx.y * blockDim.x + threadIdx.x; irow < num_irows; irow += gridDim.y * blockDim.x) {
      scalar_t *self = self_ + orow * row_size * num_irows + irow;
      scalar_t *values = values_ + orow * row_size * num_irows + irow;
      int64_t *indices = indices_ + orow * row_size * num_irows + irow;
      scalar_t out = init;
      int64_t out_idx = 0;

      for (int64_t col = 0; col < row_size; ++col) {
        if(THCNumerics<scalar_t>::isnan(*self) || (!THCNumerics<scalar_t>::isnan(out) && binary_op(*self, out))) {
          out = *self;
          out_idx = col;
        }
        *values = out;
        *indices = out_idx;
        self += num_irows;
        values += num_irows;
        indices += num_irows;
      }
    }
  }
}

template<typename scalar_t, class BinaryFunction>
__host__ void scan_outer_dim_with_indices(const Tensor& self, Tensor& values, Tensor& indices,
                                       int dim, scalar_t init, BinaryFunction binary_op) {
  int row_size = self.size(dim);
  auto sizes = self.sizes();

  // Treat all outer dimensions (i.e. dim_ < dim) as one.
  int num_orows = std::accumulate(sizes.begin(), sizes.begin() + dim, 1, std::multiplies<int>());

  // Treat all inner dimensions (i.e. dim > dimension) as one.
  int num_irows = std::accumulate(sizes.begin() + dim + 1, sizes.end(), 1, std::multiplies<int>());

  dim3 threads(std::min(512, int(num_irows)));
  int maxGridDim = at::cuda::getCurrentDeviceProperties()->maxGridSize[0];
  dim3 grid(std::min(maxGridDim, num_orows), std::min(maxGridDim, ceil_div(num_irows, int(threads.x))));

  tensor_kernel_scan_outer_dim_with_indices<scalar_t><<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    self.data_ptr<scalar_t>(), values.data_ptr<scalar_t>(), indices.data_ptr<int64_t>(),
    num_orows, num_irows, row_size, init, binary_op);
  AT_CUDA_CHECK(hipGetLastError());
}

template <typename scalar_t, class BinaryFunction>
__host__ void scan_innermost_dim_with_indices(const Tensor& self, Tensor& values, Tensor& indices, scalar_t init, BinaryFunction binary_op) {
  int ndim = self.dim();
  // Treat all outer dimensions as a single dimension.
  int row_size = self.size(ndim - 1);
  int num_rows = self.numel() / row_size;

  dim3 threads(16, 32);
  dim3 grid(std::min(at::cuda::getCurrentDeviceProperties()->maxGridSize[0], ceil_div(num_rows, int(threads.y))));

  tensor_kernel_scan_innermost_dim_with_indices<scalar_t, 16, 32><<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    self.data_ptr<scalar_t>(), values.data_ptr<scalar_t>(), indices.data_ptr<int64_t>(),
    num_rows, row_size, init, binary_op);
  AT_CUDA_CHECK(hipGetLastError());
}

template<typename scalar_t, typename BinaryFunction>
void scan_dim_with_indices(const Tensor& self, Tensor& values, Tensor& indices, //int64_t dim) {
     int64_t dim, scalar_t init, BinaryFunction binary_op) {
  int ndim = self.dim();
  Tensor self_ = self.contiguous();
  Tensor values_ = values.contiguous();
  Tensor indices_ = indices.contiguous();
   if (dim == ndim - 1) {
     scan_innermost_dim_with_indices<scalar_t>(self, values, indices, init, binary_op);
   } else {
     scan_outer_dim_with_indices<scalar_t>(self, values, indices, dim, init, binary_op);
   }
}

void cummax_helper_cuda(const Tensor& self, Tensor& values, Tensor& indices, int64_t dim) {
  TensorArg output_arg{ values, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ self, "input", 3 };
  checkAllSameGPU("cummax", {output_arg, indices_arg, input_arg});
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half,
    self.scalar_type(), "cummax_cuda", [&]() {
    scalar_t init = self.is_floating_point() ? (-1*std::numeric_limits<scalar_t>::infinity()) : std::numeric_limits<scalar_t>::lowest();
    scan_dim_with_indices<scalar_t>(self, values, indices, dim, init, std::greater_equal<scalar_t>());
  });
}

void cummin_helper_cuda(const Tensor& self, Tensor& values, Tensor& indices, int64_t dim) {
  TensorArg output_arg{ values, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ self, "input", 3 };
  checkAllSameGPU("cummin", {output_arg, indices_arg, input_arg});
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half,
    self.scalar_type(), "cummin_cuda", [&]() {
    scalar_t init = self.is_floating_point() ? std::numeric_limits<scalar_t>::infinity() : std::numeric_limits<scalar_t>::max();
    scan_dim_with_indices<scalar_t>(self, values, indices, dim, init, std::less_equal<scalar_t>());
  });
}

}} // namespace at::native
